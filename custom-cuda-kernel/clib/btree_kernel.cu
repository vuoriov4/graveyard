#include "hip/hip_runtime.h"
#include <vector>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void get_paths_kernel(
    const int* leaf_indices,
    const torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>* nodes,
    const int num_leaf_indices,
    const int num_features,
    const int max_depth,
    torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> paths) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < num_leaf_indices) {
    const int idx = leaf_indices[i];
    int c_idx = idx;
    for (int k = max_depth - 1; k >= 0; k--) {
      for (int n = 0; n < num_features; n++) {
        paths[i][k][n] = nodes[k][c_idx][n];
      }
      c_idx = c_idx / 2;
    }
  }
}

torch::Tensor get_paths_cuda(
    std::vector<int> &leaf_indices,
    std::vector<torch::Tensor> &nodes,
    int num_features,
    int max_depth) {
  int num_leaf_indices = leaf_indices.size();
  torch::Tensor paths = torch::zeros({num_leaf_indices, max_depth, num_features}).to(nodes[0].device());
  const int threads = 1024;
  const dim3 blocks(leaf_indices.size() / threads);
  AT_DISPATCH_FLOATING_TYPES(nodes[0].type(), "get_paths_cuda", ([&] {
    // Create accessor array from nodes
    std::vector<torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits>> nodes_a;
    for (int i = 0; i < max_depth; i++) {
      nodes_a.push_back(nodes[i].packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>());
    }
    // Launch kernel
    get_paths_kernel<scalar_t><<<blocks, threads>>>(
        leaf_indices.data(),
        nodes_a.data(),
        num_leaf_indices,
        num_features,
        max_depth,
        paths.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>());
  }));
  return paths;
}

